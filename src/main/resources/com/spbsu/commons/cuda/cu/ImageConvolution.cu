
#include <hip/hip_runtime.h>
extern "C"
__global__ void convolution(
        int *original,
        int *kernel,
        int *blurred,
        int kernelRadius,
        int originalWidth,
        int mirrorWidth
) {
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;

  const int kernelSize = kernelRadius * 2 + 1;

  int kernelValue = 0;
  int imageValue = 0;
  int sumR = 0;
  int sumG = 0;
  int sumB = 0;
  int r = 0;
  int g = 0;
  int b = 0;

  for(int i = 0; i <= kernelSize; i++) {
    for(int j = 0; j <= kernelSize; j++) {
      kernelValue = kernel[i * kernelSize + j];
      imageValue = original[(y + i) * mirrorWidth + (x + j)];

      r = ((imageValue >> 16) & 0xFF) * kernelValue;
      g = ((imageValue >> 8) & 0xFF) * kernelValue;
      b = ((imageValue >> 0) & 0xFF) * kernelValue;

      sumR += r;
      sumG += g;
      sumB += b;
    }
  }
  sumR = (sumR << 8) + sumG;
  sumR = (sumR << 8) + sumB;
  blurred[y * originalWidth + x] = sumR;
}