
#include <hip/hip_runtime.h>
extern "C"
__global__ void fSigmoid(float *original, int size) {
  const int X = gridDim.x;
  const int index = gridDim.y * X * threadIdx.x + X * blockIdx.y + blockIdx.x;

  if(index < size) {
    original[index] = 1.f / (1.f + expf(-original[index]));
  }
}

extern "C"
__global__ void fExp(float *original, int size) {
  const int X = gridDim.x;
  const int index = gridDim.y * X * threadIdx.x + X * blockIdx.y + blockIdx.x;

  if(index < size) {
    original[index] = expf(original[index]);
  }
}

extern "C"
__global__ void fTanh(float *original, int size) {
  const int X = gridDim.x;
  const int index = gridDim.y * X * threadIdx.x + X * blockIdx.y + blockIdx.x;

  if(index < size) {
    original[index] = tanh(original[index]);
  }
}

extern "C"
__global__ void fRndSigmoid(float *original, float *random, int size) {
  const int X = gridDim.x;
  const int index = gridDim.y * X * threadIdx.x + X * blockIdx.y + blockIdx.x;

  if(index < size) {
    original[index] = (1.f / (1.f + expf(-original[index]))) > random[index];
  }
}
